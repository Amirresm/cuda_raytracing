#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
// #include <hiprand.h>
// #include <hiprand/hiprand_kernel.h>
#include "rt_utils.cu"
#include <math.h>
#include <stdbool.h>
#include <stdio.h>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

#define cudaCheckForErrorAndSync()                                             \
  gpuErrchk(hipPeekAtLastError());                                            \
  gpuErrchk(hipDeviceSynchronize());

#define cudaStartTimer(start, stop)                                            \
  hipEventCreate(&start);                                                     \
  hipEventCreate(&stop);                                                      \
  hipEventRecord(start, 0);

#define cudaStopTimerAndRecord(start, stop, time)                              \
  hipEventRecord(stop, 0);                                                    \
  hipEventSynchronize(stop);                                                  \
  hipEventElapsedTime(&time, start, stop);

#define VP_W 0.7f
#define VP_H VP_W * 9 / 16
#define DIAFRAGM 0.01f
#define FOCAL 10
#define RAY_BOUNCE_LIMIT 10
#define RAY_COERCION_RATE 3

__device__ __forceinline__ void pixel_ray(float x, float y, Vec3 *origin,
                                          Vec3 *direction) {
  origin->x = 0;
  origin->y = 4.0f;
  origin->z = 0;

  direction->x = x;
  direction->y = y;
  direction->z = FOCAL;
  // divide_v(direction, len_v(direction));
  rotateDirection(direction, 7, 0, 0);
  normalize_v(direction);
}

__device__ __forceinline__ void trace_ray(Vec3 *origin, Vec3 *direction,
                                          int ray_count, const Object *objects,
                                          int object_count, Vec3 *ray_energy,
                                          unsigned *seed) {
  Vec3 ray_color = {.x = 1, .y = 1, .z = 1};

  Vec3 intersection, normal;
  int hit_index, reflect_count, prev_hit_index = -1;
  Vec3 r_o, r_d, emitted_light;

  Vec3 sky_color, sky_emitted_light;
  float sky_emitted_light_strength = 0.15;

  for (int i = 0; i < ray_count; i++) {
    ray_color = {.x = 1, .y = 1, .z = 1};
    reflect_count = 0;
    copy_v(&r_o, origin);
    copy_v(&r_d, direction);
    prev_hit_index = -1;
    move_point_randomly_in_circle(&r_o, seed, DIAFRAGM / 2);
    // r_d.x += my_drand(seed) * 0.005 - 0.0025;
    // r_d.y += my_drand(seed) * 0.005 - 0.0025;
    normalize_v(&r_d);
    while (reflect_count < RAY_BOUNCE_LIMIT) {
      if (find_closest_hit(&r_o, &r_d, objects, object_count, prev_hit_index,
                           &intersection, &normal, &hit_index)) {
        reflect_count++;
        const Object *obj = &objects[hit_index];
        prev_hit_index = hit_index;

        copy_v(&r_o, &intersection);
        reflect(&r_d, &normal, &r_d);
        random_direction_hemi_and_lerp(&r_d, &normal, seed,
                                       1.0 - obj->material.specular_rate);
        normalize_v(&r_d);

        emitted_light.x = obj->material.emission_color.a;
        emitted_light.y = obj->material.emission_color.b;
        emitted_light.z = obj->material.emission_color.c;
        mult_v(&emitted_light, obj->material.emission_strength);

        mult_v(&emitted_light, &ray_color);
        add_v(ray_energy, &emitted_light);

        ray_color.x *= obj->material.color.a;
        ray_color.y *= obj->material.color.b;
        ray_color.z *= obj->material.color.c;
        // float max_c = max(ray_color.x, max(ray_color.y, ray_color.z));
        // if (max_c > 1) {
        //   ray_color.x /= max_c;
        //   ray_color.y /= max_c;
        //   ray_color.z /= max_c;
        // }
      } else {
        sky_color.x = (r_d.y + 0.1) * 0.1;
        sky_color.y = (r_d.y + 0.1) * 0.5;
        sky_color.z = (r_d.y + 0.1);
        // float max_c = max(ray_energy->x, max(ray_energy->y, ray_energy->z));
        // mult_v(&sky_color, 10);
        mult_v(&sky_color, 0.01);
        sky_emitted_light.x = sky_emitted_light_strength;
        sky_emitted_light.y = sky_emitted_light_strength;
        sky_emitted_light.z = sky_emitted_light_strength;

        mult_v(&sky_emitted_light, &sky_color);

        mult_v(&sky_emitted_light, &ray_color);
        add_v(ray_energy, &sky_emitted_light);

        break;
      }
    }
  }

  ray_energy->x /= ray_count;
  ray_energy->y /= ray_count;
  ray_energy->z /= ray_count;
  // float max_e = max(ray_energy->x, max(ray_energy->y, ray_energy->z));
  // if (max_e > 1) {
  //   ray_energy->x /= max_e;
  //   ray_energy->y /= max_e;
  //   ray_energy->z /= max_e;
  // }
}

__global__ void test_kernel(const Object *objects, const int count,
                            unsigned short *r, unsigned short *g,
                            unsigned short *b, const int w, const int h,
                            const int rays, const int ray_coercion) {
  int x_p = (blockDim.x * blockIdx.x + threadIdx.x) / rays;
  int y_p = blockDim.y * blockIdx.y + threadIdx.y;

  if (x_p >= w || y_p >= h)
    return;
  int index = blockDim.x * blockIdx.x + threadIdx.x + y_p * (w * rays);
  unsigned int seed = index + 10;

  float x = ((x_p - w / 2.0) / w) * VP_W * FOCAL * 2,
        y = -((y_p - h / 2.0) / h) * VP_H * FOCAL * 2;

  Vec3 r_origin;
  Vec3 r_dir;
  pixel_ray(x, y, &r_origin, &r_dir);

  Vec3 ray_energy = {.x = 0, .y = 0, .z = 0};
  trace_ray(&r_origin, &r_dir, ray_coercion, objects, count, &ray_energy,
            &seed);

  r[index] = ray_energy.x * 255.0;
  g[index] = ray_energy.y * 255.0;
  b[index] = ray_energy.z * 255.0;
}

__global__ void average_kernel(const unsigned short *r, const unsigned short *g,
                               const unsigned short *b, UCHAR *r_out,
                               UCHAR *g_out, UCHAR *b_out, int w, int h,
                               int rays) {
  int x_p = blockDim.x * blockIdx.x + threadIdx.x;
  int y_p = blockDim.y * blockIdx.y + threadIdx.y;
  if (x_p >= w || y_p >= h)
    return;
  int index = x_p + y_p * w;

  int rp = 0;
  int gp = 0;
  int bp = 0;
  for (int i = 0; i < rays; i++) {
    int in_index = (x_p * rays) + i + y_p * (w * rays);
    rp += r[in_index];
    gp += g[in_index];
    bp += b[in_index];
  }

  r_out[index] = (rp / rays) > 255 ? 255 : (rp / rays);
  g_out[index] = (gp / rays) > 255 ? 255 : (gp / rays);
  b_out[index] = (bp / rays) > 255 ? 255 : (bp / rays);
}

void test_renderer(Scene *scene, Frame *frame, PipelineSetting setting) {
  int w = frame->width;
  int h = frame->height;
  int rays_thread_count = setting.ray_per_pixel / RAY_COERCION_RATE;
  int ray_coercion = setting.ray_per_pixel / rays_thread_count;
  unsigned short *r;
  hipMalloc(&r, sizeof(unsigned short) * w * h * rays_thread_count);
  unsigned short *g;
  hipMalloc(&g, sizeof(unsigned short) * w * h * rays_thread_count);
  unsigned short *b;
  hipMalloc(&b, sizeof(unsigned short) * w * h * rays_thread_count);

  UCHAR *r_out;
  hipMalloc(&r_out, sizeof(UCHAR) * w * h);
  UCHAR *g_out;
  hipMalloc(&g_out, sizeof(UCHAR) * w * h);
  UCHAR *b_out;
  hipMalloc(&b_out, sizeof(UCHAR) * w * h);

  Object *d_objects;
  hipMalloc(&d_objects, sizeof(Object) * scene->count);
  hipMemcpy(d_objects, scene->objects, sizeof(Object) * scene->count,
             hipMemcpyHostToDevice);

  int block_size = 16;
  dim3 thd = dim3(block_size, block_size);
  dim3 bld = dim3((w * rays_thread_count - 1) / block_size + 1,
                  (h - 1) / block_size + 1);

  printf("Width: %d, Height: %d, Rays per pixel: %d\n", w, h,
         setting.ray_per_pixel);
  printf("Grid width: %d, Grid height: %d, Total thread count: %d\n", bld.x,
         bld.y, w * h * rays_thread_count);
  printf("Total ray per pixel: %d, Thread for each ray: %d, Rays per thread: "
         "%d\n",
         rays_thread_count * ray_coercion, rays_thread_count, ray_coercion);

  float time;
  hipEvent_t start, stop;
  cudaStartTimer(start, stop);

  test_kernel<<<bld, thd>>>(d_objects, scene->count, r, g, b, w, h,
                            rays_thread_count, ray_coercion);
  cudaCheckForErrorAndSync();
  cudaStopTimerAndRecord(start, stop, time);
  printf("GPU kernel took %.4f ms \n\n", time);

  // ----
  cudaStartTimer(start, stop);

  average_kernel<<<bld, thd>>>(r, g, b, r_out, g_out, b_out, w, h,
                               rays_thread_count);
  cudaCheckForErrorAndSync();
  cudaStopTimerAndRecord(start, stop, time);
  printf("Average kernel took %.4f ms \n\n", time);

  hipMemcpy(frame->r, r_out, w * h, hipMemcpyDeviceToHost);
  hipMemcpy(frame->g, g_out, w * h, hipMemcpyDeviceToHost);
  hipMemcpy(frame->b, b_out, w * h, hipMemcpyDeviceToHost);
  hipFree(r);
  hipFree(g);
  hipFree(b);
}

int main(int argc, char *argv[]) {
  int ray_count = 10;
  int width = 1200;
  if (argc == 3) {
    width = atoi(argv[1]);
    ray_count = atoi(argv[2]);
    if (width <= 0) {
      printf("Please provide a valid positive integer for width.\n");
      return 1;
    }
    if (ray_count <= 0) {
      printf("Please provide a valid positive integer for ray_per_pixel.\n");
      return 1;
    }
  }

  int height = width * 9 / 16;
  PipelineSetting setting = {.width = width,
                             .height = height,
                             .ray_per_pixel = ray_count,
                             .debug = 1,
                             .save = 1,
                             .out_file = (char *)"test_cu.bmp"};
  Scene *scene = sample_scene_cuda();

  pipeline(scene, setting, test_renderer);

  free_scene(scene);
}
